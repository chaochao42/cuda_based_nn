#include "hip/hip_runtime.h"
#include "leaky_relu_activation.h"
#include "../nn_utils/nn_exception.h"
#include <iostream>



__global__ void LeakyReluActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim, float alpha) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) A[index] = fmaxf(Z[index], alpha * Z[index]);
}


__global__ void LeakyReluActivationBackward(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim, float alpha) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) dZ[index] = dA[index] * fmaxf((Z[index] > 0), alpha);
}

LeakyReluActivation::LeakyReluActivation(std::string name, float alpha) {
	this->name = name;
    this->alpha = alpha;
}

LeakyReluActivation::~LeakyReluActivation()
{ }

Matrix& LeakyReluActivation::forward(Matrix& Z) {
	this->Z = Z;
	A.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	LeakyReluActivationForward<<<num_of_blocks, block_size>>>(Z.data_device.get(), A.data_device.get(),
														   	Z.shape.x, Z.shape.y, this->alpha);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform reaky_relu forward propagation.");

	return A;
}

Matrix& LeakyReluActivation::backward(Matrix& dA, float learning_rate) {
	dZ.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	LeakyReluActivationBackward<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(),
															 dZ.data_device.get(),
															 Z.shape.x, Z.shape.y, this->alpha);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform reaky_relu back propagation");

	return dZ;
}
